#include <iostream>
#include <vector>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

const int N = 1000;  
vector<float> A(N), B(N), C(N);


__global__ void vectorAddGPU(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}


void CrearVectores(vector<float>& A, vector<float>& B) {
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        A[i] = (rand() % 1000) / 10.0f;
        B[i] = (rand() % 1000) / 10.0f;
    }
}


float TiempoGPU(vector<float>& A, vector<float>& B, vector<float>& C) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    auto start = std::chrono::high_resolution_clock::now();

    vectorAddGPU<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    hipMemcpy(C.data(), d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return duration.count(); 
}

int main() {
    CrearVectores(A, B);

    float time = TiempoGPU(A, B, C);

    for (int i = 0; i < 1000; i++) {
        cout << "A" << i << ": " << A[i]
             << " + B" << i << ": " << B[i]
             << " = C" << i << ": " << C[i] << endl;
    }

    cout << "El tiempo de ejecución en GPU fue: " << time << " ms" << endl;

    return 0;
}
