#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16  // Tamaño del bloque (ajustable)

int div_up(int x, int y) {
    return (x + y - 1) / y;  // Redondear hacia arriba
}

// Función Kernel que se ejecuta en el Device
__global__ void Multiplica_Matrices_GM(float *C, float *A, float *B, int nfil, int ncol) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Índice de la fila
    int idy = blockIdx.y * blockDim.y + threadIdx.y;  // Índice de la columna
    int index = idy * ncol + idx;

    if (idy < nfil && idx < ncol) {
        float sum = 0.0f;
        for (int k = 0; k < ncol; k++) {
            sum += A[idy * ncol + k] * B[k * ncol + idx];
        }
        C[index] = sum;
    }
}

int main(void) {
    // Punteros a matrices en el Host
    float *A_h, *B_h, *C_h;
    // Punteros a matrices en el Device
    float *A_d, *B_d, *C_d;
    
    int nfil = 5;  // Número de filas
    int ncol = 5;  // Número de columnas
    int N = nfil * ncol;  // Número de elementos en la matriz

    size_t size = N * sizeof(float);  // Tamaño de los arreglos

    // GPU Time
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Pedimos memoria en el Host
    A_h = (float *)malloc(size);
    B_h = (float *)malloc(size);
    C_h = (float *)malloc(size);

    // Inicializamos las matrices A y B en el Host
    for (int i = 0; i < nfil; i++) {
        for (int j = 0; j < ncol; j++) {
            A_h[i * ncol + j] = 1.0f;  // Valor inicial de A
            B_h[i * ncol + j] = 2.0f;  // Valor inicial de B
        }
    }

    // Pedimos memoria en el Device
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // Pasamos las matrices A y B del Host al Device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Realizamos el cálculo en el Device
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 n_blocks(div_up(ncol, BLOCK_SIZE), div_up(nfil, BLOCK_SIZE));
    hipEventRecord(start);
    Multiplica_Matrices_GM<<<n_blocks, block_size>>>(C_d, A_d, B_d, nfil, ncol);
    hipEventRecord(stop);
    
    // Esperamos a que termine el kernel
    hipEventSynchronize(stop);
    
    // Pasamos el resultado del Device al Host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Mostrar el resultado
    printf("\nMatriz A:\n");
    for (int i = 0; i < nfil; i++) {
        for (int j = 0; j < ncol; j++) {
            printf("%2.2f ", A_h[i * ncol + j]);
        }
        printf("\n");
    }

    printf("\nMatriz B:\n");
    for (int i = 0; i < nfil; i++) {
        for (int j = 0; j < ncol; j++) {
            printf("%2.2f ", B_h[i * ncol + j]);
        }
        printf("\n");
    }

    printf("\nMatriz C:\n");
    for (int i = 0; i < nfil; i++) {
        for (int j = 0; j < ncol; j++) {
            printf("%2.2f ", C_h[i * ncol + j]);
        }
        printf("\n");
    }

    // Calculamos el tiempo de ejecución en la GPU
    hipEventElapsedTime(&time, start, stop);
    printf("Tiempo de ejecución: %3.1f ms\n", time);

    // Liberamos la memoria del Host
    free(A_h);
    free(B_h);
    free(C_h);

    // Liberamos la memoria del Device
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}
