#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void suma_vectores(float *c, float *a, float *b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    if (idx < N) {
        c[idx] = a[idx] + b[idx];  
    }
}

int main(void) {
    t
    float *a_h, *b_h, *c_h;
    
    float *a_d, *b_d, *c_d;
    int N=10;
    size_t size = N * sizeof(float);  

    
    a_h = (float *)malloc(size);
    b_h = (float *)malloc(size);
    c_h = (float *)malloc(size);

    
    for (int i = 0; i < N; i++) {
        a_h[i] = (float)i;  
        b_h[i] = (float)(i + 1);
    }

    printf("\nArreglo a:\n");
    for (int i = 0; i < N; i++) printf("%f ", a_h[i]);
    printf("\nArreglo b:\n");
    for (int i = 0; i < N; i++) printf("%f ", b_h[i]);

    
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);

    
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    
    int block_size = 8;
    int n_blocks = (N + block_size - 1) / block_size;  
    suma_vectores<<<n_blocks, block_size>>>(c_d, a_d, b_d, N); 

    
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    
    printf("\nResultado c:\n");
    for (int i = 0; i < N; i++) printf("%f ", c_h[i]);
    printf("\n");

    
    free(a_h);
    free(b_h);
    free(c_h);

    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}
