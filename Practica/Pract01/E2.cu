#include "hip/hip_runtime.h"
__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    
    int N = 1024;
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    
    vectorAdd<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C, N);

    
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

   
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
