
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int noOfDevices;
    /* get no. of device */
    hipGetDeviceCount (&noOfDevices);

    hipDeviceProp_t prop;
    for (int i = 0; i < noOfDevices; i++)
    {
        /* get device properties */
        hipGetDeviceProperties (&prop, i);

        printf("Device Name:\t%s\n", prop.name);
        printf("Total global memory:\t%ld\n", prop.totalGlobalMem);
        printf("No. of SMs:\t%d\n", prop.multiProcessorCount);
        printf("Shared memory / SM:\t%ld\n", prop.sharedMemPerBlock);
        printf("Registers / SM:\t%d\n", prop.regsPerBlock);
    }

    return 1;
}
